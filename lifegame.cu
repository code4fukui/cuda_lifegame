#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define W 20
#define H 20
#define N (W * H)

__global__ void vector_lifegame(char* out, char* in) {
  int x0 = blockDim.x * blockIdx.x + threadIdx.x;
  int y0 = blockDim.y * blockIdx.y + threadIdx.y;
  int xp = (x0 + 1) % W;
  int xm = (x0 - 1 + W) % W;
  int yp = (y0 + 1) % H;
  int ym = (y0 - 1 + H) % W;
  int sum = 0;
  sum += in[xm + ym * W];
  sum += in[x0 + ym * W];
  sum += in[xp + ym * W];
  sum += in[xm + y0 * W];
  sum += in[xp + y0 * W];
  sum += in[xm + yp * W];
  sum += in[x0 + yp * W];
  sum += in[xp + yp * W];
  int isAlive = in[x0 + y0 * W];
  out[x0 + y0 * W] = ((isAlive && (sum == 2 || sum == 3)) || (!isAlive && sum == 3)) ? 1 : 0;
}

int main() {
  // Allocate memory
  char* in = (char*)malloc(sizeof(char) * N);
  char* out = (char*)malloc(sizeof(char) * N);

  // Initialize array
  for (int i = 0; i < N; i++){
    in[i] = 0;
  }
  in[2 + 2 * W] = 1;
  in[3 + 3 * W] = 1;
  in[1 + 4 * W] = 1;
  in[2 + 4 * W] = 1;
  in[3 + 4 * W] = 1;
  
  // Allocate device memory
  char* d_in;
  char* d_out;
  hipMalloc((void**)&d_in, sizeof(char) * N);
  hipMalloc((void**)&d_out, sizeof(char) * N);

  for (int k = 0; k < 100; k++) {
    // Transfer data from host to device memory
    hipMemcpy(d_in, in, sizeof(char) * N, hipMemcpyHostToDevice);

    // Executing kernel 
    //dim3 grid(W, H), block(1, 1); // NG
    dim3 grid(1, 1), block(W, H);
    vector_lifegame<<<grid, block>>>(d_out, d_in);

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(char) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < H; i++) {
      for (int j = 0; j < W; j++) {
        printf("%d", out[i * W + j]);
      }
      printf("\n");
    }
    printf("\n");
    // swap data
    char* t = in;
    in = out;
    out = t;
  }

  // Deallocate device memory
  hipFree(d_in);
  hipFree(d_out);

  // Deallocate host memory
  free(in);
  free(out);
}
